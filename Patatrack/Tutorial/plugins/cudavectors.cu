#include "hip/hip_runtime.h"
// system include files
#include <cmath>

// CUDA include files
#include <hip/hip_runtime.h>

// CMSSW include files
#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "cudavectors.h"

namespace cudavectors {



	  // In CUDA function type qualifiers __device__ and __host__ can be used together in which case the function is compiled for both the host and the device.
	  // https://stackoverflow.com/questions/33218522/cuda-host-device-variables
	  // https://www.geeksforgeeks.org/understanding-constexper-specifier-in-c/ 

  __host__ __device__ inline void convert(CylindricalVector const& cylindrical, CartesianVector & cartesian) {
    // fill here ...
    cartesian.x = cylindrical.rho * std::cos(cylindrical.phi);
    cartesian.y = cylindrical.rho * std::sin(cylindrical.phi);
    cartesian.z = cylindrical.rho * std::sinh(cylindrical.eta);
  }


  __global__ void convertKernel(CylindricalVector const* cylindrical, CartesianVector* cartesian, size_t size) {
    // fill here ...
    auto firstElement = threadIdx.x + blockIdx.x * blockDim.x;
    auto gridSize = blockDim.x * gridDim.x;

    for (size_t i = firstElement; i < size; i += gridSize) {
      convert(cylindrical[i], cartesian[i]);
    }// for loop
  }// funct



  void convertWrapper(CylindricalVector const* cylindrical, CartesianVector* cartesian, size_t size) {
    // fill here ...
    // grid contains blocks, each of which holds threads
    // blocks share the fast elements of memory
    // number of blocks of "grid" is basically the same thing
    auto blockSize = 512;                                // somewhat arbitrary
    auto gridSize = (size + blockSize - 1) / blockSize;  // round up to cover the sample size; this is how MANY BLOCKS we want

    convertKernel<<<gridSize, blockSize>>>(cylindrical, cartesian, size);
    cudaCheck(hipGetLastError());

  }

}  // namespace cudavectors
